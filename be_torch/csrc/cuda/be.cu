#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace be_torch {

__global__ void be_kernel(const long int* edge_index, long int N, long int* result) {
  //int idx = blockIdx.x * blockDim.x + threadIdx.x;
  //if (idx < numel) result[idx] = a[idx] * b[idx] + c;
}

at::Tensor be_cuda(const at::Tensor& edge_index, long int N) {
  //TORCH_CHECK(edge_index.sizes()[0] == 2);
  TORCH_INTERNAL_ASSERT(edge_index.device().type() == at::DeviceType::CUDA);
  at::Tensor edge_index_contig = edge_index_contig.contiguous();
  at::Tensor result = torch::empty(N, edge_index_contig.options());
  const float* edge_index_ptr = a_contig.data_ptr<long int>();
  float* result_ptr = result.data_ptr<long int>();

  be_kernel<<<(N+255)/256, 256>>>(edge_index_ptr, N, result_ptr);
  return result;
}

// Registers CUDA implementations for mymuladd, mymul, myadd_out
TORCH_LIBRARY_IMPL(be_torch, CUDA, m) {
  m.impl("be", &be_cuda);
}

}
